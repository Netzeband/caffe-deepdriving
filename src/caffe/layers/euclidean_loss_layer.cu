#include <vector>

#include "caffe/layers/euclidean_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  
  // DeepDriving Changes: Normalize the training data before calculating the loss
  //
  // For any reason, the original DeepDriving training data is not normalized, thus
  // we have to do it here before calculating the loss. This is very... ugly!
  // Maybe it would be better to implement a script which preprocess the training
  // data by normalizing it before training starts. I will think about it in a further
  // implementation.
  
  int BatchSize = bottom[0]->num();   
  int OutputDimension = count/BatchSize;
  
  const Dtype* CudaResult = bottom[0]->gpu_data();
  const Dtype* CudaLabel  = bottom[1]->gpu_data();
  
  Dtype Label[BatchSize*14];
  Dtype Result[count]; 
  Dtype LabelArray[count];
  
  hipMemcpy(Result, CudaResult, sizeof(Dtype) * count,          hipMemcpyDeviceToHost);
  hipMemcpy(Label,  CudaLabel,  sizeof(Dtype) * BatchSize * 14, hipMemcpyDeviceToHost);
    
  for (int i = 0; i < BatchSize; ++i) 
  {
    LabelArray[i * OutputDimension]      = Label[i*14+0]/1.1+0.5;     // angle range ~ [-0.5, 0.5]
    if (LabelArray[i * OutputDimension]>1.0) LabelArray[i * OutputDimension]=1.0;
    if (LabelArray[i * OutputDimension]<0.0) LabelArray[i * OutputDimension]=0.0;

    LabelArray[i * OutputDimension + 1]  = Label[i*14+1]*0.17778+1.34445;   // toMarking_L range ~ [-7, -2.5]
    LabelArray[i * OutputDimension + 2]  = Label[i*14+2]*0.14545+0.39091;   // toMarking_M range ~ [-2, 3.5]
    LabelArray[i * OutputDimension + 3]  = Label[i*14+3]*0.17778-0.34445;   // toMarking_R range ~ [2.5, 7]
    LabelArray[i * OutputDimension + 4]  = Label[i*14+4]/95.0+0.12;   // dist_L range ~ [0, 75]
    LabelArray[i * OutputDimension + 5]  = Label[i*14+5]/95.0+0.12;   // dist_R range ~ [0, 75]
    LabelArray[i * OutputDimension + 6]  = Label[i*14+6]*0.14545+1.48181;   // toMarking_LL range ~ [-9.5, -4]
    LabelArray[i * OutputDimension + 7]  = Label[i*14+7]*0.16+0.98;   // toMarking_ML range ~ [-5.5, -0.5]
    LabelArray[i * OutputDimension + 8]  = Label[i*14+8]*0.16+0.02;   // toMarking_MR range ~ [0.5, 5.5]
    LabelArray[i * OutputDimension + 9]  = Label[i*14+9]*0.14545-0.48181;   // toMarking_RR range ~ [4, 9.5]
    LabelArray[i * OutputDimension + 10] = Label[i*14+10]/95.0+0.12;   // dist_LL range ~ [0, 75]
    LabelArray[i * OutputDimension + 11] = Label[i*14+11]/95.0+0.12;   // dist_MM range ~ [0, 75]
    LabelArray[i * OutputDimension + 12] = Label[i*14+12]/95.0+0.12;   // dist_RR range ~ [0, 75]
    LabelArray[i * OutputDimension + 13] = Label[i*14+13]*0.6+0.2;   // fast range ~ {0, 1}
  }
  
  Dtype* CudaLabelArray;
  hipMalloc((void**)&CudaLabelArray,    sizeof(Dtype) * count);
  hipMemcpy(CudaLabelArray, LabelArray, sizeof(Dtype) * count, hipMemcpyHostToDevice);
  
  caffe_gpu_sub(
      count,
      CudaResult,
      CudaLabelArray,
      diff_.mutable_gpu_data());
      
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;

  hipFree(CudaLabelArray);
  
  /*
  Dtype Differences[count]; 
  hipMemcpy(Differences, diff_.gpu_data(), sizeof(Dtype) * count, hipMemcpyDeviceToHost);
  
  //for (int i = 0; i < BatchSize; ++i) 
  {
    int i=0;
    for (int j = 0; j < OutputDimension; ++j) 
    {
      printf("BatchElement: %d, Value: %d, Result: %f, LabelArray: %f, diff: %f \n", 
      	i, 
      	j, 
      	Result[i*OutputDimension+j], 
      	LabelArray[i*OutputDimension+j], 
      	Differences[i*OutputDimension+j]); 
      fflush(stdout);
    }    
  }
  printf("Current Loss: %f\n", loss);
  */
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe
